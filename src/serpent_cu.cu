#include "hip/hip_runtime.h"
// See serpent.h for legal information.

#include "serpent.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifndef SUBKEY_LENGTH
#define SUBKEY_LENGTH 132
#endif


/*!	\brief Decrypt a single block on the device.
 */
__device__ void serpent_cuda_decrypt_block(block128_t* block, uint32_t* subkey);


/*!	\brief Decrypt the specified array of blocks with the specified subkey through a CUDA thread.
 */
__global__ void serpent_cuda_decrypt_blocks(block128_t* cuda_blocks);


/*!	\brief Encrypt a single block on the device.
 */
__device__ void serpent_cuda_encrypt_block(block128_t* block, uint32_t* subkey);


/*!	\brief Encrypt the specified array of blocks with the specified subkey through a CUDA thread.
 */
__global__ void serpent_cuda_encrypt_blocks(block128_t* cuda_blocks);


/*!	\brief Flip the bytes of the specified 32-bit unsigned integer.
 *	\note Tried to make a global function for this but got
 *		"Error: External calls are not supported...".
 *
 *	\return	A 32-bit unsigned integer with the bytes mirrored.
 */
__device__ uint32_t serpent_mirror_bytes32(uint32_t x);


// Constant variables must be declared with a static scope...
// Some variables are prefixed with the file name because of
// "duplicate global variable looked up by string name" errors.
//! Array to hold the expanded serpent key.
__device__ __constant__ uint32_t cuda_subkey[SUBKEY_LENGTH];
//! The total number of blocks being decrypted by a single CUDA thread.
__device__ __constant__ int serpent_blocks_per_thread;
//! The total number of blocks being decrypted in the entire CUDA kernel.
__device__ __constant__ int serpent_blocks_per_kernel;


extern "C"
inline int serpent_cuda_allocate_buffer(size_t free_global_memory, size_t total_global_memory, int block_count, int multiprocessor_count, int thread_count, block128_t** cuda_blocks, size_t* used_global_memory_output, int* blocks_per_kernel_output, int* blocks_per_thread_output, int* buffer_allocation_attempts_output) {
	hipError_t cuda_error;
	int blocks_per_kernel;
	int blocks_per_thread;
	int buffer_allocation_attempts;

	// Validate parameters.
	#ifdef DEBUG_SERPENT
	if ( free_global_memory < 1 ) {
		fprintf(stderr, "Free global memory less than 1.\n");
		return -1;
	}
	else if ( cuda_blocks == NULL ) {
		fprintf(stderr, "cuda_blocks was NULL.\n");
		return -1;
	}
	else if ( used_global_memory_output == NULL ) {
		fprintf(stderr, "used_global_memory_output was NULL.\n");
		return -1;
	}
	else if ( blocks_per_kernel_output == NULL ) {
		fprintf(stderr, "blocks_per_kernel_output was NULL.\n");
		return -1;
	}
	else if ( blocks_per_thread_output == NULL ) {
		fprintf(stderr, "blocks_per_thread_output was NULL.\n");
		return -1;
	}
	#endif

	// Try to allocate blocks.
	buffer_allocation_attempts = 1;
	while ( true ) {
		// Subtract a small portion of global memory.
		free_global_memory -= total_global_memory * SERPENT_CUDA_MEMORY_MULTIPLIER;
		if ( free_global_memory <= 0 ) {
			fprintf(stderr, "No memory for blocks available.\n");
			return -1;
		}

		// Calculate number of blocks per thread.
		blocks_per_kernel = free_global_memory / sizeof(block128_t);
		if ( blocks_per_kernel > block_count ) {
			blocks_per_kernel = block_count;
		}
		blocks_per_thread = (blocks_per_kernel / multiprocessor_count) / thread_count;

		// Attempt to allocate memory on the GPU.
		cuda_error = hipMalloc( (void**)cuda_blocks, (int)(sizeof(block128_t) * blocks_per_kernel) );
		if ( cuda_error == hipSuccess ) { // Success! Exit this loop.
			break;
		}
		else if ( cuda_error != hipErrorOutOfMemory ) {
			fprintf(stderr, "Unable to malloc blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}

		buffer_allocation_attempts++;
	}

	// Assign output parameters.
	(*used_global_memory_output) = free_global_memory;
	(*blocks_per_kernel_output) = blocks_per_kernel;
	(*blocks_per_thread_output) = blocks_per_thread;
	if ( buffer_allocation_attempts_output != NULL ) {
		(*buffer_allocation_attempts_output) = buffer_allocation_attempts;
	}

	// Return success.
	return 0;
}


__device__ void serpent_cuda_decrypt_block(block128_t* block, uint32_t* subkey) {
	uint32_t a, b, c, d, e;
	int j;

	// Change to little endian.
	a = serpent_mirror_bytes32(block->x0);
	b = serpent_mirror_bytes32(block->x1);
	c = serpent_mirror_bytes32(block->x2);
	d = serpent_mirror_bytes32(block->x3);

	// Decrypt the current block.
	j = 4;
	subkey += 96;
	beforeI7(KX);
	goto start;
	do
	{
		c = b;
		b = d;
		d = e;
		subkey -= 32;
		beforeI7(inverse_linear_transformation);
	start:
		beforeI7(I7); afterI7(KX);
		afterI7(inverse_linear_transformation); afterI7(I6); afterI6(KX);
		afterI6(inverse_linear_transformation); afterI6(I5); afterI5(KX);
		afterI5(inverse_linear_transformation); afterI5(I4); afterI4(KX);
		afterI4(inverse_linear_transformation); afterI4(I3); afterI3(KX);
		afterI3(inverse_linear_transformation); afterI3(I2); afterI2(KX);
		afterI2(inverse_linear_transformation); afterI2(I1); afterI1(KX);
		afterI1(inverse_linear_transformation); afterI1(I0); afterI0(KX);
	}
	while (--j != 0);

	// Restore to big endian based on algorithm-defined order.
	block->x0 = serpent_mirror_bytes32(a);
	block->x1 = serpent_mirror_bytes32(d);
	block->x2 = serpent_mirror_bytes32(b);
	block->x3 = serpent_mirror_bytes32(e);
}


__global__ void serpent_cuda_decrypt_blocks(block128_t* cuda_blocks) {
	int index = (blockIdx.x * (blockDim.x * serpent_blocks_per_thread)) + threadIdx.x;
	int i;

	// Decrypt the minimal number of blocks.
	for ( i = 0; i < serpent_blocks_per_thread; i++ ) {
		serpent_cuda_decrypt_block(&(cuda_blocks[index]), cuda_subkey);

		index += blockDim.x;
	}

	// Decrypt the extra blocks that fall outside the minimal number of blocks.
	index = ( gridDim.x * blockDim.x * serpent_blocks_per_thread ) + ((blockIdx.x * blockDim.x) + threadIdx.x); // (end of array) + (absolute thread #).
	if ( index < serpent_blocks_per_kernel ) {
		serpent_cuda_decrypt_block(&(cuda_blocks[index]), cuda_subkey);
	}
}

/*
__global__ void serpent_cuda_decrypt_blocks(block128_t* cuda_blocks) {
	int index = (blockIdx.x * blockDim.x * blocks_per_thread) + (threadIdx.x * blocks_per_thread); // (beginning of multiprocessor segment) + (segment index).
	int i;

	// Encrypted the minimal number of blocks.
	for ( i = 0; i < blocks_per_thread; i++ ) {
		serpent_cuda_decrypt_block(&(cuda_blocks[index + i]), cuda_subkey);
	}

	// Encrypt the extra blocks that fall outside the minimal number of block.s
	index = ( gridDim.x * blockDim.x * blocks_per_thread ) + ((blockIdx.x * blockDim.x) + threadIdx.x); // (end of array) + (absolute thread #).
	if ( index < blocks_per_kernel ) {
		serpent_cuda_decrypt_block(&(cuda_blocks[index]), cuda_subkey);
	}
}
*/

__device__ void serpent_cuda_encrypt_block(block128_t* block, uint32_t* subkey) {
	uint32_t a, b, c, d, e;
	int j;

	// Change to little endian.
	a = serpent_mirror_bytes32(block->x0);
	b = serpent_mirror_bytes32(block->x1);
	c = serpent_mirror_bytes32(block->x2);
	d = serpent_mirror_bytes32(block->x3);

	// Encrypt the current block.
	j = 1;
	do {
		beforeS0(KX); beforeS0(S0); afterS0(linear_transformation);
		afterS0(KX); afterS0(S1); afterS1(linear_transformation);
		afterS1(KX); afterS1(S2); afterS2(linear_transformation);
		afterS2(KX); afterS2(S3); afterS3(linear_transformation);
		afterS3(KX); afterS3(S4); afterS4(linear_transformation);
		afterS4(KX); afterS4(S5); afterS5(linear_transformation);
		afterS5(KX); afterS5(S6); afterS6(linear_transformation);
		afterS6(KX); afterS6(S7);

		if (j == 4)
			break;

		++j;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		subkey += 32;
		beforeS0(linear_transformation);
	} while (1);
	afterS7(KX);

	// Restore to big endian based on algorithm-defined order.
	block->x0 = serpent_mirror_bytes32(d);
	block->x1 = serpent_mirror_bytes32(e);
	block->x2 = serpent_mirror_bytes32(b);
	block->x3 = serpent_mirror_bytes32(a);
}


__global__ void serpent_cuda_encrypt_blocks( block128_t* cuda_blocks ) {
	int index = (blockIdx.x * (blockDim.x * serpent_blocks_per_thread)) + threadIdx.x;
	int i;

	// Encrypt the minimal number of blocks.
	for ( i = 0; i < serpent_blocks_per_thread; i++ ) {
		// Encrypt the block.
		serpent_cuda_encrypt_block(&(cuda_blocks[index]), cuda_subkey);

		// Increment the index.
		index += blockDim.x;
	}

	// Encrypt the extra blocks that fall outside the minimal number of block.s
	index = (gridDim.x * (blockDim.x * serpent_blocks_per_thread)) + ((blockIdx.x * blockDim.x) + threadIdx.x); // (end of array) + (absolute thread #).
	if ( index < serpent_blocks_per_kernel ) {
		serpent_cuda_encrypt_block(&(cuda_blocks[index]), cuda_subkey);
	}
}

/* A better attempt at stronger global memory coalescing. Still did not turn out well.
#define UINT32_PER_BLOCK128 4

__device__ void serpent_cuda_encrypt_block(uint32_t* shared_blocks, int shared_index, uint32_t* subkey);
__global__ void serpent_cuda_encrypt_blocks(uint32_t* cuda_blocks);

__global__ void serpent_cuda_encrypt_blocks( uint32_t* cuda_blocks ) {
	int threads_per_multiprocessor = blockDim.x;
	int cache_index = (blockIdx.x * (UINT32_PER_BLOCK128 * threads_per_multiprocessor * blocks_per_thread));
	int i;

	// Encrypt the minimal number of blocks.
	for ( i = 0; i < blocks_per_thread; i++ ) {
		// Encrypt the blocks at the cache index
		serpent_cuda_encrypt_block(&(cuda_blocks[cache_index]), cuda_subkey, threads_per_multiprocessor);

		// Adjust cache index value.
		cache_index += (threads_per_multiprocessor * UINT32_PER_BLOCK128);
	}

	// Encrypt the extra blocks that fall outside the minimal number of block.
	// NOTE: DOES NOT WORK and is incomplete.
	//cache_index = (gridDim.x * (threads_per_multiprocessor * blocks_per_thread)) + (blockIdx.x * threads_per_multiprocessor); // (end of array + multiprocessor block).
	//if ( cache_index > blocks_per_kernel) {
	//	return;
	//}
	//else if ( (cache_index + threads_per_multiprocessor) > blocks_per_kernel ) {
	//	if ( threadIdx.x + cache_index > blocks_per_kernel ) {
	//		return;
	//	}
	//	threads_per_multiprocessor = (blocks_per_kernel - cache_index);
	//}
}
__device__ void serpent_cuda_encrypt_block(uint32_t* global_blocks, uint32_t* subkey, int threads_per_multiprocessor) {
	// Array that allows collaborative loading of blocks into shared memory.
	extern __shared__ uint32_t shared_blocks[];
	uint32_t a, b, c, d, e;
	int index = threadIdx.x;
	int j;

	// Collaboratively load blocks into shared memory.
	shared_blocks[index] = serpent_mirror_bytes32(global_blocks[index]);
	index += threads_per_multiprocessor;
	shared_blocks[index] = serpent_mirror_bytes32(global_blocks[index]);
	index += threads_per_multiprocessor;
	shared_blocks[index] = serpent_mirror_bytes32(global_blocks[index]);
	index += threads_per_multiprocessor;
	shared_blocks[index] = serpent_mirror_bytes32(global_blocks[index]);
	index -= (threads_per_multiprocessor * 3);
	__syncthreads();

	// Read from shared memory.
	index *= UINT32_PER_BLOCK128;
	a = shared_blocks[index];
	b = shared_blocks[index+1];
	c = shared_blocks[index+2];
	d = shared_blocks[index+3];

	// Encrypt the current block.
	j = 1;
	do {
		beforeS0(KX); beforeS0(S0); afterS0(linear_transformation);
		afterS0(KX); afterS0(S1); afterS1(linear_transformation);
		afterS1(KX); afterS1(S2); afterS2(linear_transformation);
		afterS2(KX); afterS2(S3); afterS3(linear_transformation);
		afterS3(KX); afterS3(S4); afterS4(linear_transformation);
		afterS4(KX); afterS4(S5); afterS5(linear_transformation);
		afterS5(KX); afterS5(S6); afterS6(linear_transformation);
		afterS6(KX); afterS6(S7);

		if (j == 4)
			break;

		++j;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		subkey += 32;
		beforeS0(linear_transformation);
	} while (1);
	afterS7(KX);

	// Write blocks back to global memory.
	global_blocks[index] = serpent_mirror_bytes32(d);
	global_blocks[index+1] = serpent_mirror_bytes32(e);
	global_blocks[index+2] = serpent_mirror_bytes32(b);
	global_blocks[index+3] = serpent_mirror_bytes32(a);
	__syncthreads();
}

serpent_cuda_encrypt_blocks<<<multiprocessor_count, thread_count, (sizeof(block128_t) * thread_count)>>>((uint32_t*)cuda_blocks);

 */

__device__ uint32_t serpent_mirror_bytes32(uint32_t x) {
	uint32_t out;

	// Change to Little Endian.
	out = (uint8_t) x;
       	out <<= 8; out |= (uint8_t) (x >> 8);
	out <<= 8; out |= (uint8_t) (x >> 16);
	out = (out << 8) | (uint8_t) (x >> 24);

	// Return out.
	return out;
}


extern "C"
int serpent_cuda_decrypt_cu(uint32_t* subkey, block128_t* blocks, int block_count, size_t* buffer_size) {
	// Total number of registers taken up by a single CUDA thread.
	const int REGISTERS_PER_THREAD = 8;
	block128_t* cuda_blocks;
	hipError_t cuda_error;
	size_t total_global_memory;
	size_t free_global_memory;
	int blocks_per_kernel;
	int blocks_per_thread;
	int buffer_allocation_attempts;
	int count; 
	int device_number;
	int kernel_invocation_attempts;
	int multiprocessor_count;
	int temp;
	int thread_count;
	int i;

	// Validate parameters.
	#ifdef DEBUG_SERPENT
	if ( subkey == NULL ) {
		fprintf(stderr, "subkey was NULL.\n");
		return -1;
	}
	else if ( blocks == NULL ) {
		fprintf(stderr, "blocks was NULL.\n");
		return -1;
	}
	else if ( block_count < 1 ) {
		fprintf(stderr, "block_count was less than 1.\n");
		return -1;
	}
	else if ( buffer_size == NULL ) {
		fprintf(stderr, "buffer_size was NULL.\n");
		return -1;
	}
	#endif

	// Get the number of devices.
	cuda_error = hipGetDeviceCount( &count );
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get device count: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}
	else if ( count == 0 ) {
		fprintf(stderr, "No CUDA-capable devices found.\n");
		return -1;
	}

	// Calculate multiprocessor and thread count.
	device_number = 0;
	if ( cuda_get_block_and_thread_count_max(device_number, REGISTERS_PER_THREAD, &multiprocessor_count, &thread_count) == -1 ) {
		fprintf(stderr, "Unable to get multiprocessor and thread count.\n");
		return -1;
	}

	// Move subkey to constant memory.
	cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "cuda_subkey"), subkey, sizeof(uint32_t) * SUBKEY_LENGTH);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to copy subkey to constant memory: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Calculate the amount of global memory available for blocks.
	cuda_error = hipMemGetInfo(&free_global_memory, &total_global_memory);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get memory information: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Allocate blocks buffer on the GPU.
	if ( serpent_cuda_allocate_buffer(free_global_memory, total_global_memory, block_count, multiprocessor_count, thread_count, &cuda_blocks, &free_global_memory, &blocks_per_kernel, &blocks_per_thread, &buffer_allocation_attempts) == -1 ) {
		fprintf(stderr, "Unable to allocate initial buffer.\n");
		return -1;
	}

	// Decrypt the blocks.
	i = 0;
	while (i < block_count) {
		// Corner case.
		if ( i + blocks_per_kernel > block_count ) {
			blocks_per_kernel = block_count - i;
			blocks_per_thread = blocks_per_kernel / multiprocessor_count / thread_count;
		}

		// Run the algorithm.
		kernel_invocation_attempts = 1;
		while( true ) {
			// Move blocks to global memory.
			cuda_error = hipMemcpy( cuda_blocks, &(blocks[i]), sizeof(block128_t) * blocks_per_kernel, hipMemcpyHostToDevice );
			if ( cuda_error != hipSuccess ) {
				fprintf(stderr, "Unable to memcopy blocks: %s.\n", hipGetErrorString(cuda_error));
				return -1;
			}

			// Copy blocks per thread to constant memory.
			cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "serpent_blocks_per_thread"), &blocks_per_thread, sizeof(int));
			if ( cuda_error != hipSuccess ) {
				fprintf(stderr, "Unable to copy blocks_per_thread to constant memory: %s.\n", hipGetErrorString(cuda_error));
				return -1;
			}

			// Copy blocks per kernel to constant memory.
			cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "serpent_blocks_per_kernel"), &blocks_per_kernel, sizeof(int));
			if ( cuda_error != hipSuccess ) {
				fprintf(stderr, "Unable to copy blocks_per_kernel to constant memory: %s.\n", hipGetErrorString(cuda_error));
				return -1;
			}

			// Run encryption.
			serpent_cuda_decrypt_blocks<<<multiprocessor_count, thread_count>>>(cuda_blocks);
			cuda_error = hipGetLastError();
			if ( cuda_error == hipSuccess ) { // Successful run.
				break;
			}
			else if ( cuda_error != hipErrorOutOfMemory ) { // Unexpected error.
				fprintf(stderr, "ERROR invoking the kernel: %s, %i.\n", hipGetErrorString(cuda_error), cuda_error);
				return -1;
			}

			// Free the old blocks buffer.
			hipFree(cuda_blocks);

			// Allocate a new blocks buffer.
			if ( serpent_cuda_allocate_buffer(free_global_memory, total_global_memory, block_count, multiprocessor_count, thread_count, &cuda_blocks, &free_global_memory, &blocks_per_kernel, &blocks_per_thread, &temp) == -1 ) {
				fprintf(stderr, "Unable to reallocate blocks buffer.\n");
				return -1;
			}
			buffer_allocation_attempts += temp;

			kernel_invocation_attempts++;
		}

		// Get blocks from global memory.
		cuda_error = hipMemcpy( &(blocks[i]), cuda_blocks, sizeof(block128_t) * blocks_per_kernel, hipMemcpyDeviceToHost );
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to retrieve blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}
	
		// Increment i by the number of blocks processed.
		i += blocks_per_kernel;
	}

	// Free blocks from global memory.
	hipFree(cuda_blocks);

	// Assign output parameters.
	(*buffer_size) = free_global_memory;

	// TODO: Add these as output parameters.
	//fprintf(stderr, "Buffer allocation attempts: %i.\nKernel invocation attempts: %i.\n",
	//	buffer_allocation_attempts, kernel_invocation_attempts);

	// Return success.
	return 0;
}


extern "C"
int serpent_cuda_encrypt_cu(uint32_t* subkey, block128_t* blocks, int block_count, size_t* buffer_size) {
	// Maximum total number of registers taken up by a single CUDA thread.
	// This variable will need to be manually calculated and updated if
	// the algorithm implementation changes (but if you know of a way
	// to proceedurally do this, please, feel free...).
	const int REGISTERS_PER_THREAD = 8;
	//hipDeviceProp_t cuda_device;
	block128_t* cuda_blocks;
	hipError_t cuda_error;
	size_t total_global_memory;
	size_t free_global_memory;
	int buffer_allocation_attempts;
	int kernel_invocation_attempts;
	int blocks_per_kernel;
	int blocks_per_thread;
	int count;
	int device_number;
	int multiprocessor_count;
	int temp;
	int thread_count;
	int i;

	// Validate parameters.
	#ifdef DEBUG_SERPENT
	if ( subkey == NULL ) {
		fprintf(stderr, "subkey was NULL.\n");
		return -1;
	}
	else if ( blocks == NULL ) {
		fprintf(stderr, "blocks was NULL.\n");
		return -1;
	}
	else if ( block_count < 1 ) {
		fprintf(stderr, "block_count was less than 1.\n");
		return -1;
	}
	else if ( buffer_size == NULL ) {
		fprintf(stderr, "buffer_size was NULL.\n");
		return -1;
	}
	#endif

	// Get the number of devices.
	cuda_error = hipGetDeviceCount( &count );
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get device count: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}
	else if ( count == 0 ) {
		fprintf(stderr, "No CUDA-capable devices found.\n");
		return -1;
	}

	// Calculate the number of multiprocessors and threads to launch.
	device_number = 0;
	if ( cuda_get_block_and_thread_count_max(device_number, REGISTERS_PER_THREAD, &multiprocessor_count, &thread_count) == -1 ) {
		fprintf(stderr, "Unable to get max thread count.\n");
		return -1;
	}

	// Copy the subkey to constant memory.
	cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "cuda_subkey"), subkey, sizeof(uint32_t) * SUBKEY_LENGTH);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to copy subkey to constant memory: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Calculate the amount of global memory available for blocks.
	cuda_error = hipMemGetInfo(&free_global_memory, &total_global_memory);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get memory information: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Allocate blocks buffer on the GPU.
	if ( serpent_cuda_allocate_buffer(free_global_memory, total_global_memory, block_count, multiprocessor_count, thread_count, &cuda_blocks, &free_global_memory, &blocks_per_kernel, &blocks_per_thread, &buffer_allocation_attempts) == -1 ) {
		fprintf(stderr, "Unable to allocate initial buffer.\n");
		return -1;
	}
	
	// Encrypt the blocks.
	i = 0;
	while (i < block_count) {
		// Corner case.
		if ( i + blocks_per_kernel > block_count ) {
			blocks_per_kernel = block_count - i;
			blocks_per_thread = blocks_per_kernel / multiprocessor_count / thread_count;
		}

		// Run the algorithm.
		kernel_invocation_attempts = 1;
		while ( true ) {
			// Move blocks to global memory.
			cuda_error = hipMemcpy( cuda_blocks, &(blocks[i]), sizeof(block128_t) * blocks_per_kernel, hipMemcpyHostToDevice );
			if ( cuda_error != hipSuccess ) {
				fprintf(stderr, "Unable to memcopy blocks: %s.\n", hipGetErrorString(cuda_error));
				return -1;
			}

			// Copy blocks per thread to constant memory.
			cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "serpent_blocks_per_thread"), &blocks_per_thread, sizeof(int));
			if ( cuda_error != hipSuccess ) {
				fprintf(stderr, "Unable to copy blocks_per_thread to constant memory: %s.\n", hipGetErrorString(cuda_error));
				return -1;
			}

			// Copy blocks per kernel to constant memory.
			cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "serpent_blocks_per_kernel"), &blocks_per_kernel, sizeof(int));
			if ( cuda_error != hipSuccess ) {
				fprintf(stderr, "Unable to copy blocks_per_kernel to constant memory: %s.\n", hipGetErrorString(cuda_error));
				return -1;
			}

			// Run encryption.
			serpent_cuda_encrypt_blocks<<<multiprocessor_count, thread_count>>>(cuda_blocks);
			cuda_error = hipGetLastError();
			if ( cuda_error == hipSuccess ) { // Successful run.
				break;
			}
			else if ( cuda_error != hipErrorOutOfMemory ) { // Unexpected error.
				fprintf(stderr, "ERROR invoking the kernel: %s, %i.\n", hipGetErrorString(cuda_error), cuda_error);
				return -1;
			}

			// Free the old blocks buffer.
			hipFree(cuda_blocks);

			// Allocate a new blocks buffer.
			if ( serpent_cuda_allocate_buffer(free_global_memory, total_global_memory, block_count, multiprocessor_count, thread_count, &cuda_blocks, &free_global_memory, &blocks_per_kernel, &blocks_per_thread, &temp) == -1 ) {
				fprintf(stderr, "Unable to reallocate blocks buffer.\n");
				return -1;
			}
			buffer_allocation_attempts += temp;

			kernel_invocation_attempts++;
		}
		
		// Get blocks from global memory.
		cuda_error = hipMemcpy( &(blocks[i]), cuda_blocks, sizeof(block128_t) * blocks_per_kernel, hipMemcpyDeviceToHost );
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to retrieve blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}
	
		// Increment i by the number of blocks processed.
		i += blocks_per_kernel;
	}

	// Free blocks from global memory.
	hipFree(cuda_blocks);

	// Assign output parameters.
	(*buffer_size) = free_global_memory;

	// TODO: Makes this as function output.
	//fprintf(stderr, "Buffer allocation attempts: %i.\nKernel invocation attempts: %i.\n",
		//buffer_allocation_attempts, kernel_invocation_attempts);

	// Return success.
	return 0;
}
